#include "hip/hip_runtime.h"
#include "src/gpu.cuh"
#include "src/utils.hpp"

template <typename Dtype>
__global__ void sum(Dtype *a, Dtype *b, Dtype *c, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i <= N) {
    c[i] = a[i] + b[i];
  }
}

template <typename Dtype>
void AddGPUKernel(Dtype *in_a, Dtype *in_b, Dtype *out_c, int N,
                  hipStream_t stream) {
  sum<Dtype>
      <<<GET_BLOCKS(N), CUDA_NUM_THREADS, 0, stream>>>(in_a, in_b, out_c, N);

  hipError_t err = hipGetLastError();
  if (hipSuccess != err)
    throw std::runtime_error(Formatter()
                             << "CUDA kernel failed : " << std::to_string(err));
}

template void AddGPUKernel<float>(float *in_a, float *in_b, float *out_c, int N,
                                  hipStream_t stream);
